#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <omp.h>

#define BLOCK_FACTOR 32
#define NUM_THREAD 1024
#define PART (BLOCK_FACTOR * BLOCK_FACTOR) / NUM_THREAD

const int INF = ((1 << 30) - 1);
void input(char* inFileName);
void output(char* outFileName);
void printArray(int* Dist);

void block_FW();
int ceil(int a, int b);

const int V = 40010;
int n, m;

int numDevs = 0;
int* Dist;
size_t pitch;

clock_t begin, end;
double IO_time = 0;
double kernel_time = 0;

int main(int argc, char* argv[]) {
    hipGetDeviceCount(&numDevs);
    printf("devices count: %d\n", numDevs);

    hipMallocManaged(&Dist, V * V * sizeof(int));

    begin = clock();
    input(argv[1]);
    end = clock();
    IO_time += (double) (end - begin) / CLOCKS_PER_SEC;
    //printArray(Dist);

    begin = clock();
    block_FW();
    end = clock();
    kernel_time += (double) (end - begin) / CLOCKS_PER_SEC;
    //printArray(Dist);

    begin = clock();
    output(argv[2]);
    end = clock();
    IO_time += (double) (end - begin) / CLOCKS_PER_SEC;

    hipFree(&Dist);

    printf("I/O time: %f secs.\n", IO_time);
    printf("GPU kernel time: %f secs.\n", kernel_time);
    return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    printf("V = %d, E = %d\n", n, m);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }
    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i * n + j] >= INF)
                Dist[i * n + j] = INF;
            fwrite(&Dist[i * n + j], sizeof(int), 1, outfile);
        }
    }
    fclose(outfile);
}

void printArray(int* Dist){
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            printf("%6d ", Dist[i * n + j]);
        }
        printf("\n");
    } 
}

int ceil(int a, int b) { return (a + b - 1) / b; }

/*
__device__ void block_cal(int* C, int* A, int* B, int j, int i) {
    for (int k = 0; k < BLOCK_FACTOR; k++) {
        int sum = A[i*BLOCK_FACTOR + k] + B[k*BLOCK_FACTOR + j];
        if (C[i*BLOCK_FACTOR + j] > sum) {
            C[i*BLOCK_FACTOR + j] = sum;
        }
        __syncthreads();
    }
}
*/

__global__ void cal_phase1(int Round, int block_start_x, int block_start_y, int* Dist, int n) {
    int x = threadIdx.y;
    int y = threadIdx.x;
    int i = (block_start_x + blockIdx.x) * BLOCK_FACTOR + x;
    int j = (block_start_y + blockIdx.y) * BLOCK_FACTOR + y;

    __shared__ int C[BLOCK_FACTOR][BLOCK_FACTOR+1];
    int i_start = i * n;
    int rb = Round * BLOCK_FACTOR;
    C[x][y] = Dist[i_start + j];
    __syncthreads();

    if(i >= n || j >= n) return;

    for (int k = 0; k < BLOCK_FACTOR && rb + k < n; k++) {
        int sum = C[x][k] + C[k][y];
        if (sum < C[x][y])
            C[x][y] = sum;
        __syncthreads();
    }
    Dist[i_start + j] = C[x][y];
}

__global__ void cal_phase2A(int Round, int block_start_x, int block_start_y, int* Dist, int n) {
    int x = threadIdx.y;
    int y = threadIdx.x;
    int i = (block_start_x + blockIdx.x) * BLOCK_FACTOR + x;
    int j = (block_start_y + blockIdx.y) * BLOCK_FACTOR + y;

    __shared__ int C[BLOCK_FACTOR][BLOCK_FACTOR+1];
    __shared__ int A[BLOCK_FACTOR][BLOCK_FACTOR+1];
    int i_start = i * n;
    int rb = Round * BLOCK_FACTOR;
    C[x][y] = Dist[i_start + j];
    A[x][y] = Dist[i_start + rb + y];
    __syncthreads();

    if(i >= n || j >= n) return;
    
    for (int k = 0; k < BLOCK_FACTOR && rb + k < n; k++) {
        int sum = A[x][k] + C[k][y];
        if (sum < C[x][y])
            C[x][y] = sum;
        __syncthreads();
    }
    Dist[i_start + j] = C[x][y];
}


__global__ void cal_phase2B(int Round, int block_start_x, int block_start_y, int* Dist, int n) {
    int x = threadIdx.y;
    int y = threadIdx.x;
    int i = (block_start_x + blockIdx.x) * BLOCK_FACTOR + x;
    int j = (block_start_y + blockIdx.y) * BLOCK_FACTOR + y;

    __shared__ int C[BLOCK_FACTOR][BLOCK_FACTOR+1];
    __shared__ int B[BLOCK_FACTOR][BLOCK_FACTOR+1];
    int i_start = i * n;
    int rb = Round * BLOCK_FACTOR;
    C[x][y] = Dist[i_start + j];
    B[x][y] = Dist[((rb + x) * n) + j];
    __syncthreads();

    if(i >= n || j >= n) return;

    for (int k = 0; k < BLOCK_FACTOR && rb + k < n; k++) {
        int sum = C[x][k] + B[k][y];
        if (sum < C[x][y])
            C[x][y] = sum;
        __syncthreads();
    }
    Dist[i_start + j] = C[x][y];
}

__global__ void cal_phase3(int Round, int block_start_x, int block_start_y, int* Dist, int n) {
    int nn = n;    
    int x = threadIdx.y;
    int y = threadIdx.x;
    int i = (block_start_x + blockIdx.x) * BLOCK_FACTOR + x;
    int j = (block_start_y + blockIdx.y) * BLOCK_FACTOR + y;

    __shared__ int A[BLOCK_FACTOR][BLOCK_FACTOR+1];
    __shared__ int B[BLOCK_FACTOR][BLOCK_FACTOR+1];
    int i_start = i * nn;
    int rb = Round * BLOCK_FACTOR;
    int c = Dist[i_start + j];
    A[x][y] = Dist[i_start + rb + y];
    B[x][y] = Dist[((rb + x) * nn) + j];
    __syncthreads();

    if(i >= nn || j >= nn) return;

    for (int k = 0; k < BLOCK_FACTOR && rb + k < nn; k++) {
        int sum = A[x][k] + B[k][y];
        if (sum < c) 
            c = sum;
    }
    Dist[i_start + j] = c;
}

void block_FW() {
    int round = ceil(n, BLOCK_FACTOR);
    dim3 block_dim(BLOCK_FACTOR, BLOCK_FACTOR);
    dim3 grid;
    hipDeviceSynchronize();
    for (int r = 0; r < round; ++r) {
        //printf("%d %d\n", r, round);
        //fflush(stdout);
        int bs = round - r - 1;
        hipDeviceSynchronize();
        
        /* phase 1 */
        hipSetDevice(0);
        grid.x = 1; 
        grid.y = 1;
        cal_phase1 <<<grid, block_dim >>> (r, r, r, Dist, n);
        hipDeviceSynchronize();
        /* phase 2 */
        if (r > 0) {
            grid.x = 1;
            grid.y = r;
            cal_phase2A <<<grid, block_dim>>> (r, r, 0, Dist, n);
        }
        if (bs > 0) {
            grid.x = 1;
            grid.y = bs;
            cal_phase2A <<<grid, block_dim>>> (r, r, r + 1, Dist, n);
        }
        if (r > 0) {
            grid.x = r;
            grid.y = 1;
            cal_phase2B <<<grid, block_dim>>> (r, 0, r, Dist, n);
        }
        if (bs > 0) {
            grid.x = bs;
            grid.y = 1;
            cal_phase2B <<<grid, block_dim>>> (r, r + 1, r, Dist, n);
        }
        hipDeviceSynchronize();
        /* phase 3 */
#pragma omp parallel num_threads(2)
{
        int tid = omp_get_thread_num();
        hipSetDevice(tid);
        if (tid == 0) {
        if (r > 0) {
            grid.x = r;
            grid.y = r;
            cal_phase3 <<<grid, block_dim>>> (r, 0, 0, Dist, n);
            hipDeviceSynchronize();
        }
        }
        if (tid == 1) {
        if (r > 0 && bs > 0) {
            grid.x = r;
            grid.y = bs;
            cal_phase3 <<<grid, block_dim>>> (r, 0, r + 1, Dist, n);
            hipDeviceSynchronize();
            
            grid.x = bs;
            grid.y = r;
            cal_phase3 <<<grid, block_dim>>> (r, r + 1, 0, Dist, n);
            hipDeviceSynchronize();
        }
        }
        if (tid == 0) {
        if (bs > 0) {
            grid.x = bs;
            grid.y = bs;
            cal_phase3 <<<grid, block_dim>>> (r, r + 1, r + 1, Dist, n);
            hipDeviceSynchronize();
        }
        }
}
    }
    hipDeviceSynchronize();
}






















